#include "hip/hip_runtime.h"
#include "Array.hpp"

namespace LinPack
{

template <typename T>
__global__ void _Fill(size_t n, T val, T *ptr)
{
	size_t thread_idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (thread_idx >= n) return;
	ptr[thread_idx] = val;
}

template <typename T>
void Array_t<T, is_device_t<T>>::Fill(const_reference val)
{
	dim3 threads(1024, 1, 1);
	dim3 blocks(device_size() / threads.x + 1, 1, 1);

	_Fill <<< blocks, threads >>> (device_size(), val, device_ptr());

	MyBase::Fill(val);
}

template class Array_t<bool>;
template class Array_t<char>;
template class Array_t<int>;
template class Array_t<int2>;
template class Array_t<int3>;
template class Array_t<int4>;
template class Array_t<float>;
template class Array_t<float2>;
template class Array_t<float3>;
template class Array_t<float4>;
template class Array_t<double>;
template class Array_t<double2>;
template class Array_t<double3>;
template class Array_t<double4>;

}